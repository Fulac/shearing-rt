#include "hip/hip_runtime.h"
// C headers
#include <cstdio>
#include <cmath>

// C++ headers
#include <string>

// Boost headers
#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/ini_parser.hpp>

// Local headers
#include "cmplx.h"
#include "fft.h"
#include "fourier.h"
#include "fields.h"
#include "time_integral.h"
#include "shear.h"

#define FILENAMELEN 256

/* ---------------------------------------------------------------------------------------------- */
/*  Global Variables Definition                                                                   */
/* ---------------------------------------------------------------------------------------------- */

// プログラム全体で使用する変数を定義
bool   write_fields;
int    nwrite;
cureal output_time, next_output_time;

// このファイル内でのみ使用するグローバル変数を定義
namespace{
    FILE *fp;
    char *filename;
    int  *kx_index;
    cucmplx *aomgz, *aphi, *arho;
    cureal  *ensp_ao_kx, *ensp_ap_kx, *ensp_ar_kx;
    cureal  *ensp_ao_ky, *ensp_ap_ky, *ensp_ar_ky;
}

////////////////////////////////////////////////////////////////////////////////////////////////////


/* ---------------------------------------------------------------------------------------------- */
/*  Function Prototype                                                                            */
/* ---------------------------------------------------------------------------------------------- */

void input_data
    ( void
);

template <class T>
T readEntry
   ( boost::property_tree::ptree pt
   , std::string section
   , std::string name
   , T defaultValue
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void init_output
    ( void
);

void finish_output
    ( void
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void output_fields
    ( const int    istep
    , const cureal time
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void output_maxamp
    ( const cureal time
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void en_spectral
    ( const int    istep
    , const cureal time
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void ks_reim
    ( const cureal time
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void k_data_bef
    ( const cureal time
    , const int    istep
);

void k_data_aft
    ( const cureal time
    , const int    istep
);

////////////////////////////////////////////////////////////////////////////////////////////////////


/* ---------------------------------------------------------------------------------------------- */
/*  Function Definition                                                                           */
/* ---------------------------------------------------------------------------------------------- */

void input_data
    ( void
){
    boost::property_tree::ptree pt;

    try{
        boost::property_tree::read_ini( "config.ini", pt );
    }
    catch( std::exception &e ){
        printf( "ERROR: unable to read config file: %s", e.what() );
        exit(1);
    }

    // simulation parameters
    nx      = readEntry<int>( pt, "simulation", "nx",  512 );
    ny      = readEntry<int>( pt, "simulation", "ny", 1024 );
    nthread = readEntry<int>( pt, "simulation", "cuda thread num", 1024 );
    Lx      = readEntry<cureal>( pt, "simulation", "Lx", M_PI );
    Ly      = readEntry<cureal>( pt, "simulation", "Ly", M_PI );
    delt    = readEntry<cureal>( pt, "simulation", "time step", 1e-3 );
    tmax    = readEntry<cureal>( pt, "simulation", "time max", 30 );
    cfl_num = readEntry<cureal>( pt, "simulation", "cfl number", 1e-1 );
    
    /* Lx *= 2; */
    /* Ly *= 2; */

    // output parameters
    output_time  = readEntry<cureal>( pt, "output", "output time step",  1.0  );
    nwrite       = readEntry<int>(    pt, "output", "output loop count", 100  );
    write_fields = readEntry<bool>(   pt, "output", "write output",      true );

    // problem parameters
    noise_flag  = readEntry<bool>(   pt, "problem", "initial noise", true  );
    linear_flag = readEntry<bool>(   pt, "problem", "linear eq",    false );
    nu          = readEntry<cureal>( pt, "problem", "nu",            1e-3  );
    kappa       = readEntry<cureal>( pt, "problem", "kappa",         1e-5  );
    sigma       = readEntry<cureal>( pt, "problem", "sigma",         1.0   );
    g           = readEntry<cureal>( pt, "problem", "g",             1.0   );
    rho0_prime  = readEntry<cureal>( pt, "problem", "rho0_prime",    1.0   );
    rho0        = readEntry<cureal>( pt, "problem", "rho0",          1.0   );
    rho_eps1    = readEntry<cureal>( pt, "problem", "rho_eps1",      1e-2  );
    rho_eps2    = 0.1 * rho_eps1;
}

template <class T>
T readEntry
   ( boost::property_tree::ptree pt
   , std::string section
   , std::string name
   , T           defaultValue
){
   T value;

   try {
      // get value
      value = pt.get<T>( section+"."+name );
   }
   catch( boost::property_tree::ptree_error &err ) {
      // show warning if key is missing
      printf( "WARNING: readEntry: Key \"%s\" in section [%s] not found. Using default.\n"
            , name.c_str(), section.c_str() );
      value = defaultValue;
   }

   return value;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void init_output
    ( void 
){
    next_output_time = output_time;
    filename = new char[FILENAMELEN];

    aomgz = new cucmplx[nkx*nky];
    aphi  = new cucmplx[nkx*nky];
    arho  = new cucmplx[nkx*nky];

    ensp_ao_kx = new cureal[nkx];
    ensp_ap_kx = new cureal[nkx];
    ensp_ar_kx = new cureal[nkx];

    ensp_ao_ky = new cureal[nky];
    ensp_ap_ky = new cureal[nky];
    ensp_ar_ky = new cureal[nky];

    kx_index = new int[nkx];
    for( int ikx = nkxh; ikx < nkx; ikx++ ) kx_index[ikx-nkxh] = ikx;
    for( int ikx = 0; ikx < nkxh; ikx++ ) kx_index[ikx+(nkx-nkxh)] = ikx;
}

void finish_output
    ( void
){
    delete[] filename;
    delete[] kx_index;

    delete[] aomgz;
    delete[] aphi;
    delete[] arho;

    delete[] ensp_ao_kx;
    delete[] ensp_ap_kx;
    delete[] ensp_ar_kx;

    delete[] ensp_ao_ky;
    delete[] ensp_ap_ky;
    delete[] ensp_ar_ky;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void output_fields
    ( const int    istep
    , const cureal time 
){
    dim3 block( nthread );
    dim3 rgrid( (nx*ny+nthread-1)/nthread );
    dim3 cgrid( (nkx*nky+nthread-1)/nthread );

    printf("time = %g\n", time);

    if( sigma ){
        ktox_shear( dv_aomg0, dv_omgz );
        ktox_shear( dv_aphi,  dv_phi );
        ktox_shear( dv_arho0, dv_rho );
    }
    else{
        ktox( dv_aomg0, dv_omgz );
        ktox( dv_aphi,  dv_phi );
        ktox( dv_arho0, dv_rho );
    }

    hipMemcpy( omgz, dv_omgz, sizeof(cureal)*nx*ny, hipMemcpyDeviceToHost );
    hipMemcpy( phi,  dv_phi,  sizeof(cureal)*nx*ny, hipMemcpyDeviceToHost );
    hipMemcpy( rho,  dv_rho,  sizeof(cureal)*nx*ny, hipMemcpyDeviceToHost );

    snprintf( filename, FILENAMELEN, "n%05d_t%09.6f.dat", istep/nwrite, time );
    if( (fp=fopen(filename, "w+")) == NULL ) exit(1);
    for( int ix = 0; ix <= nx; ix++ ){
        for( int iy = 0; iy <= ny; iy++ ){
            if( ix < nx && iy < ny ){
                fprintf( fp, "%+e %+e %+e %+e %+e\n"
                       , xx[ix], yy[iy], omgz[ix*ny+iy], phi[ix*ny+iy], rho[ix*ny+iy] );
            }
            else if( ix == nx && iy == ny ){
                fprintf( fp, "%+e %+e %+e %+e %+e\n"
                       , xx[ix], yy[iy], omgz[0], phi[0], rho[0] );
            }
            else if( ix == nx ){
                fprintf( fp, "%+e %+e %+e %+e %+e\n"
                       , xx[ix], yy[iy], omgz[iy], phi[iy], rho[iy] );
            }
            else if( iy == ny ){
                fprintf( fp, "%+e %+e %+e %+e %+e\n"
                       , xx[ix], yy[iy], omgz[ix*ny], phi[ix*ny], rho[ix*ny] );
            }
        }
        fprintf( fp, "\n" );
    }
    fclose( fp );
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void output_maxamp
    ( const cureal time
){
    if( sigma ){
        ktox_shear( dv_aphi,  dv_phi );
        ktox_shear( dv_arho0, dv_rho );
    }
    else{
        ktox( dv_aphi,  dv_phi );
        ktox( dv_arho0, dv_rho );
    }

    snprintf( filename, FILENAMELEN, "phi_hat.dat" );
    if( (fp=fopen(filename, "a+")) == NULL ) exit(1);
    fprintf( fp, "%.10f %+e\n", time, maxvalue_search(dv_phi) );
    fclose( fp );

    snprintf( filename, FILENAMELEN, "rho_hat.dat" );
    if( (fp=fopen(filename, "a+")) == NULL ) exit(1);
    fprintf( fp, "%.10f %+e\n", time, maxvalue_search(dv_rho) );
    fclose( fp );
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void en_spectral
    ( const int    /*istep*/
    , const cureal time
){
    cureal re, im;

    if( time == 0 ){
        snprintf( filename, FILENAMELEN, "kx_ensp.txt" );
        snprintf( filename, FILENAMELEN, "ky_ensp.txt" );
    }
    else{
        hipMemcpy( aomgz, dv_aomg0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );
        hipMemcpy( aphi,  dv_aphi,  sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );
        hipMemcpy( arho,  dv_arho0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );

        snprintf( filename, FILENAMELEN, "kx_ensp_t%09.6f.dat", time );
        if( (fp=fopen(filename, "w+")) == NULL ) exit(1);

        for( int ikx = 0; ikx < nkx; ikx++ ){
            for( int iky = 0; iky < nky; iky++ ){
                if( iky == 0 ){
                    re = aomgz[ikx*nky+iky].x;
                    im = aomgz[ikx*nky+iky].y;
                    ensp_ao_kx[ikx] = re*re + im*im;

                    re = aphi[ikx*nky+iky].x;
                    im = aphi[ikx*nky+iky].y;
                    ensp_ap_kx[ikx] = re*re + im*im; 

                    re = arho[ikx*nky+iky].x;
                    im = arho[ikx*nky+iky].y;
                    ensp_ar_kx[ikx] = re*re + im*im; 
                }
                else{
                    re  = aomgz[ikx*nky+iky].x;
                    im  = aomgz[ikx*nky+iky].y;
                    ensp_ao_kx[ikx] += re*re + im*im;

                    re  = aphi[ikx*nky+iky].x;
                    im  = aphi[ikx*nky+iky].y;
                    ensp_ap_kx[ikx] += re*re + im*im; 

                    re  = arho[ikx*nky+iky].x;
                    im  = arho[ikx*nky+iky].y;
                    ensp_ar_kx[ikx] += re*re + im*im; 
                }
            }
        }
        for( int ikx = 0; ikx < nkx; ikx++ ) 
            fprintf( fp, "%+e %+e %+e %+e\n", 
                     ky[ikx], ensp_ao_kx[ikx]/nky, ensp_ap_kx[ikx]/nky, ensp_ar_kx[ikx]/nky );
        fclose( fp );

        snprintf( filename, FILENAMELEN, "ky_ensp_t%09.6f.dat", time );
        if( (fp=fopen(filename, "w+")) == NULL ) exit(1);

        for( int ikx = 0; ikx < nkx; ikx++ ){
            for( int iky = 0; iky < nky; iky++ ){
                if( ikx == 0 ){
                    re = aomgz[ikx*nky+iky].x;
                    im = aomgz[ikx*nky+iky].y;
                    ensp_ao_ky[iky] = re*re + im*im;

                    re = aphi[ikx*nky+iky].x;
                    im = aphi[ikx*nky+iky].y;
                    ensp_ap_ky[iky] = re*re + im*im; 

                    re = arho[ikx*nky+iky].x;
                    im = arho[ikx*nky+iky].y;
                    ensp_ar_ky[iky] = re*re + im*im; 
                }
                else{
                    re  = aomgz[ikx*nky+iky].x;
                    im  = aomgz[ikx*nky+iky].y;
                    ensp_ao_ky[iky] += re*re + im*im;

                    re  = aphi[ikx*nky+iky].x;
                    im  = aphi[ikx*nky+iky].y;
                    ensp_ap_ky[iky] += re*re + im*im; 

                    re  = arho[ikx*nky+iky].x;
                    im  = arho[ikx*nky+iky].y;
                    ensp_ar_ky[iky] += re*re + im*im; 
                }
            }
        }
        for( int iky = 0; iky < nky; iky++ ) 
            fprintf( fp, "%+e %+e %+e %+e\n", 
                     ky[iky], ensp_ao_ky[iky]/nkx, ensp_ap_ky[iky]/nkx, ensp_ar_ky[iky]/nkx );
        fclose( fp );
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void ks_reim
    ( const cureal time
){
    cureal ao, ap, ar;

    hipMemcpy( aomgz, dv_aomg0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );
    hipMemcpy( aphi,  dv_aphi,  sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );
    hipMemcpy( arho,  dv_arho0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );

    snprintf( filename, FILENAMELEN, "ksre_t%09.6f.dat", time );
    if( (fp=fopen(filename, "w+")) == NULL ) exit(1);
    for( int ikx = 0; ikx < nkx; ikx++ ){
        for( int iky = 0; iky < nky; iky++ ){
            ao = fabs( aomgz[kx_index[ikx]*nky+iky].x ); 
            ap = fabs( aphi[kx_index[ikx]*nky+iky].x ); 
            ar = fabs( arho[kx_index[ikx]*nky+iky].x ); 

            fprintf( fp, "%+e %+e %+e %+e %+e\n",
                     kx[kx_index[ikx]], ky[iky], ao, ap, ar );
        }
        fprintf( fp, "\n" );
    }
    fclose( fp );

    snprintf( filename, FILENAMELEN, "ksim_t%09.6f.dat", time );
    if( (fp=fopen(filename, "w+")) == NULL ) exit(1);
    for( int ikx = 0; ikx < nkx; ikx++ ){
        for( int iky = 0; iky < nky; iky++ ){
            ao = fabs( aomgz[kx_index[ikx]*nky+iky].y ); 
            ap = fabs( aphi[kx_index[ikx]*nky+iky].y ); 
            ar = fabs( arho[kx_index[ikx]*nky+iky].y ); 

            fprintf( fp, "%+e %+e %+e %+e %+e\n",
                     kx[kx_index[ikx]], ky[iky], ao, ap, ar );
        }
        fprintf( fp, "\n" );
    }
    fclose( fp );
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void k_data_bef
    ( const cureal time
    , const int    istep
){
    hipMemcpy( aomgz, dv_aomg0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );
    hipMemcpy( aphi,  dv_aphi,  sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );
    hipMemcpy( arho,  dv_arho0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );

    snprintf( filename, FILENAMELEN, "befk_n%05d_t%09.6f.dat", istep/nwrite, time );
    if( (fp=fopen(filename, "w+")) == NULL ) exit(1);

    fprintf( fp, "///////////////////////////// omg //////////////////////////////\n");
    for( int ikx = 0; ikx < nkx; ikx++ ){
        for( int iky = 0; iky < nky; iky++ ){
            fprintf( fp, "%+e ", aomgz[ikx*nky+iky].x );
        }
        fprintf( fp, "\n" );
    }
    fprintf( fp, "\n" );
    fprintf( fp, "\n" );
    for( int ikx = 0; ikx < nkx; ikx++ ){
        for( int iky = 0; iky < nky; iky++ ){
            fprintf( fp, "%+e ", aomgz[ikx*nky+iky].y );
        }
        fprintf( fp, "\n" );
    }
    fprintf( fp, "///////////////////////////////////////////////////////////////\n\n");

    fprintf( fp, "///////////////////////////// phi //////////////////////////////\n");
    for( int ikx = 0; ikx < nkx; ikx++ ){
        for( int iky = 0; iky < nky; iky++ ){
            fprintf( fp, "%+e ", aphi[ikx*nky+iky].x );
        }
        fprintf( fp, "\n" );
    }
    fprintf( fp, "\n" );
    fprintf( fp, "\n" );
    for( int ikx = 0; ikx < nkx; ikx++ ){
        for( int iky = 0; iky < nky; iky++ ){
            fprintf( fp, "%+e ", aphi[ikx*nky+iky].y );
        }
        fprintf( fp, "\n" );
    }
    fprintf( fp, "///////////////////////////////////////////////////////////////\n\n");

    fprintf( fp, "///////////////////////////// rho //////////////////////////////\n");
    for( int ikx = 0; ikx < nkx; ikx++ ){
        for( int iky = 0; iky < nky; iky++ ){
            fprintf( fp, "%+e ", arho[ikx*nky+iky].x );
        }
        fprintf( fp, "\n" );
    }
    fprintf( fp, "\n" );
    fprintf( fp, "\n" );
    for( int ikx = 0; ikx < nkx; ikx++ ){
        for( int iky = 0; iky < nky; iky++ ){
            fprintf( fp, "%+e ", arho[ikx*nky+iky].y );
        }
        fprintf( fp, "\n" );
    }
    fprintf( fp, "///////////////////////////////////////////////////////////////\n");

    fclose( fp );
}

void k_data_aft
    ( const cureal time
    , const int    istep
){
    if( time == 0 ){
        snprintf( filename, FILENAMELEN, "after_kdata.txt" );
    }
    else{
        hipMemcpy( aomgz, dv_aomg0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );
        hipMemcpy( aphi,  dv_aphi,  sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );
        hipMemcpy( arho,  dv_arho0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToHost );

        snprintf( filename, FILENAMELEN, "aftk_n%05d_t%09.6f.dat", istep/nwrite, time );
        if( (fp=fopen(filename, "w+")) == NULL ) exit(1);

        fprintf( fp, "///////////////////////////// omg //////////////////////////////\n");
        for( int ikx = 0; ikx < nkx; ikx++ ){
            for( int iky = 0; iky < nky; iky++ ){
                fprintf( fp, "%+e ", aomgz[ikx*nky+iky].x );
            }
            fprintf( fp, "\n" );
        }
        fprintf( fp, "\n" );
        fprintf( fp, "\n" );
        for( int ikx = 0; ikx < nkx; ikx++ ){
            for( int iky = 0; iky < nky; iky++ ){
                fprintf( fp, "%+e ", aomgz[ikx*nky+iky].y );
            }
            fprintf( fp, "\n" );
        }
        fprintf( fp, "///////////////////////////////////////////////////////////////\n\n");

        fprintf( fp, "///////////////////////////// phi //////////////////////////////\n");
        for( int ikx = 0; ikx < nkx; ikx++ ){
            for( int iky = 0; iky < nky; iky++ ){
                fprintf( fp, "%+e ", aphi[ikx*nky+iky].x );
            }
            fprintf( fp, "\n" );
        }
        fprintf( fp, "\n" );
        fprintf( fp, "\n" );
        for( int ikx = 0; ikx < nkx; ikx++ ){
            for( int iky = 0; iky < nky; iky++ ){
                fprintf( fp, "%+e ", aphi[ikx*nky+iky].y );
            }
            fprintf( fp, "\n" );
        }
        fprintf( fp, "///////////////////////////////////////////////////////////////\n\n");

        fprintf( fp, "///////////////////////////// rho //////////////////////////////\n");
        for( int ikx = 0; ikx < nkx; ikx++ ){
            for( int iky = 0; iky < nky; iky++ ){
                fprintf( fp, "%+e ", arho[ikx*nky+iky].x );
            }
            fprintf( fp, "\n" );
        }
        fprintf( fp, "\n" );
        fprintf( fp, "\n" );
        for( int ikx = 0; ikx < nkx; ikx++ ){
            for( int iky = 0; iky < nky; iky++ ){
                fprintf( fp, "%+e ", arho[ikx*nky+iky].y );
            }
            fprintf( fp, "\n" );
        }
        fprintf( fp, "///////////////////////////////////////////////////////////////\n");

        fclose( fp );
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
