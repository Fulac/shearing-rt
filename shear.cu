#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include "cmplx.h"
#include "fft.h"
#include "four.h"
#include "fields.h"

namespace{
    int *ikx_indexed, *dv_ikx_indexed, jump_flag, *dv_jump;
    __device__ int *gb_ikx_indexed, *gb_jump;
    cureal *dv_ky_shift, *kperp2_shear, *dv_kperp2_shear;

    cureal  *dv_rtmp;
    cucmplx *dv_ctmp1, *dv_ctmp2, *dv_ctmp3;
}

__device__ cureal *gb_ky_shift, *gb_kperp2_shear;

/* ---------------------------------------------------------------- */

void init_shear( void );
void finish_shear( void );

__global__ void ddy_shear( const cucmplx*, cucmplx* );
__global__ void laplacian_shear( const cucmplx*, cucmplx* );
__global__ void neg_lapinv_shear( const cucmplx*, cucmplx* );
void get_vector_shear( const cucmplx*, cureal*, cureal* );
void poisson_bracket_shear( const cureal*, const cureal*, const cucmplx*, cureal* );

__global__ void seq_ktox_shear( const cucmplx*, cureal* );
void ktox_shear( const cucmplx*, cureal* );
__global__ static void idft_shear_y( const cucmplx*, cucmplx* );
__global__ static void pad2d( const cucmplx*, cucmplx* );

void update_shear( const cureal );
__global__ static void shearing_ky( const cureal );
__global__ static void get_kperp2_shear( void );
__global__ static void shearing_field( const cucmplx*, cucmplx* );

/* ---------------------------------------------------------------- */

void init_shear
    ( void
){
    kperp2_shear = new cureal [nkx*nky];

    hipMalloc( (void**)&dv_kperp2_shear, sizeof(cureal)*nkx*nky );
    hipMemcpy( dv_kperp2_shear, kperp2, sizeof(cureal)*nkx*nky, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( gb_kperp2_shear), &dv_kperp2_shear, sizeof(dv_kperp2_shear) );

    ikx_indexed = new int [nkx];
    ikx_indexed[0] = 0;
    for( int ikx = 1; ikx < nkx; ikx++ ) ikx_indexed[ikx] = nkx - ikx;

    hipMalloc( (void**)&dv_ikx_indexed, sizeof(int) * nkx );
    hipMemcpy( dv_ikx_indexed, ikx_indexed, sizeof(int) * nkx, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( gb_ikx_indexed), &dv_ikx_indexed, sizeof(dv_ikx_indexed) );

    hipMalloc( (void**)&dv_jump, sizeof(int)*nkx );
    hipMemset( dv_jump, 0, sizeof(int)*nkx );
    hipMemcpyToSymbol(HIP_SYMBOL( gb_jump), &dv_jump, sizeof(dv_jump) );

    hipMalloc( (void**)&dv_ky_shift, sizeof(cureal)*nkx );
    hipMemset( dv_ky_shift, 0, sizeof(cureal)*nkx );
    hipMemcpyToSymbol(HIP_SYMBOL( gb_ky_shift), &dv_ky_shift, sizeof(dv_ky_shift) );

    hipMalloc( (void**)&dv_rtmp,  sizeof(cureal)*nx*ny );
    hipMalloc( (void**)&dv_ctmp1, sizeof(cucmplx)*nkx*nky );
    hipMalloc( (void**)&dv_ctmp2, sizeof(cucmplx)*nkx*ny );
    hipMalloc( (void**)&dv_ctmp3, sizeof(cucmplx)*nx*ny );
}

void finish_shear
    ( void
){
    delete[] kperp2_shear;
    delete[] ikx_indexed;

    hipFree( dv_jump );
    hipFree( dv_ky_shift );
    hipFree( dv_kperp2_shear );
    hipFree( dv_rtmp );
    hipFree( dv_ctmp1 );
    hipFree( dv_ctmp2 );
    hipFree( dv_ctmp3 );
}

__global__ void ddy_shear
    ( const cucmplx *in
    ,       cucmplx *out
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_nky, yid = tid%ct_nky;
    cureal kys;

    if( tid < ct_nkx*ct_nky ){
        kys = gb_ky[yid] + gb_ky_shift[xid];
        out[tid] = CIMAG * kys * in[tid];
    }
}

__global__ void laplacian_shear
    ( const cucmplx *in
    ,       cucmplx *out
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if( tid < ct_nkx*ct_nky ) out[tid] = -gb_kperp2_shear[tid] * in[tid];
}

__global__ void neg_lapinv_shear
    ( const cucmplx *in
    ,       cucmplx *out
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if( tid <= ct_nkx*ct_nky-1 ) out[tid] = in[tid] / gb_kperp2_shear[tid];
}

void get_vector_shear
    ( const cucmplx *dv_aphi
    ,       cureal  *dv_vectx
    ,       cureal  *dv_vecty
){
    dim3 block( nthread );
    dim3 cgrid( (nkx*nky+nthread-1)/nthread );

    ddy_shear <<< cgrid, block >>> ( dv_aphi, dv_ctmp1 );
    ktox( dv_ctmp1, dv_vectx );

    neg_ddx <<< cgrid, block >>> ( dv_aphi, dv_ctmp1 );
    ktox( dv_ctmp1, dv_vecty );
}

void poisson_bracket_shear
    ( const cureal  *dv_vectx
    , const cureal  *dv_vecty
    , const cucmplx *in
    ,       cureal  *out
){
    dim3 block( nthread );
    dim3 rgrid( (nx*ny+nthread-1)/nthread );
    dim3 cgrid( (nkx*nky+nthread-1)/nthread );

    ddx <<< cgrid, block >>> ( in, dv_ctmp1 );
    ktox( dv_ctmp1, out );
    mult_real_field <<< rgrid, block >>> ( dv_vectx, out );

    ddy_shear <<< cgrid, block >>> ( in, dv_ctmp1 );
    ktox( dv_ctmp1, dv_rtmp );
    mult_real_field <<< rgrid, block >>> ( dv_vecty, dv_rtmp );

    add_real_field <<< rgrid, block >>> ( dv_rtmp, out );
}

__global__ void seq_ktox_shear
    ( const cucmplx *in
    ,       cureal  *out
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_ny, yid = tid%ct_ny;

    if( tid < ct_nx*ct_ny ){
        cureal kx, ky;
        cureal out_data = 0;

        for( int ikx = 0; ikx < ct_nkx; ikx++ ){
            kx = gb_kx[ikx];
            ky = gb_ky[0] + gb_ky_shift[ikx];
            out_data += in[ikx*ct_nky].x*cos(kx*gb_xx[xid]+ky*gb_yy[yid])
                      - in[ikx*ct_nky].y*sin(kx*gb_xx[xid]+ky*gb_yy[yid]);

            for( int iky = 1; iky < ct_nky; iky++ ){
                ky = gb_ky[iky] + gb_ky_shift[ikx];
                out_data += 2*in[ikx*ct_nky+iky].x*cos(kx*gb_xx[xid]+ky*gb_yy[yid])
                          - 2*in[ikx*ct_nky+iky].y*sin(kx*gb_xx[xid]+ky*gb_yy[yid]);
            }
        }
        out[tid] = out_data;
    }
}

void ktox_shear
    ( const cucmplx *in
    ,       cureal  *out
){
    dim3 block( nthread );
    dim3 rcgrid( (nkx*ny+nthread-1)/nthread );
    dim3 rgrid( (nx*ny+nthread-1)/nthread );

    idft_shear_y <<< rcgrid, block >>> ( in, dv_ctmp2 );
    pad2d <<< rgrid, block >>> ( dv_ctmp2, dv_ctmp3 );
    ktox_1d( dv_ctmp3, out );
}

__global__ static void idft_shear_y
    ( const cucmplx *in
    ,       cucmplx *out
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_ny, yid = tid%ct_ny;

    if( tid < ct_nkx*ct_ny ){
        cucmplx out_data = CZERO;
        cureal  y_data   = gb_yy[yid];
        cureal  ky;
        cureal  ky_shift = gb_ky_shift[xid];

        ky = gb_ky[0] + ky_shift;

        out_data.x = in[xid*ct_nky].x * cos( ky * y_data )
                   - in[xid*ct_nky].y * sin( ky * y_data );

        out_data.y = in[xid*ct_nky].y * cos( ky * y_data )
                   + in[xid*ct_nky].x * sin( ky * y_data );

        for( int iky = 1; iky < ct_nky; iky++ ){
            ky = gb_ky[iky] + ky_shift;

            out_data.x += 2 * in[xid*ct_nky+iky].x * cos( ky * y_data )
                        - 2 * in[xid*ct_nky+iky].y * sin( ky * y_data );

            out_data.y += 2 * in[xid*ct_nky+iky].y * cos( ky * y_data )
                        + 2 * in[xid*ct_nky+iky].x * sin( ky * y_data );
        }

        out[tid] = out_data;
    }
}

__global__ static void pad2d
    ( const cucmplx *in
    ,       cucmplx *out
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_ny, yid = tid%ct_ny;

    if( tid < ct_nx*ct_ny ){
        if( xid < ct_nkxh )       out[tid] = in[xid*ct_ny+yid];
        else if( xid > ct_nkxh2 ) out[tid] = in[(xid-ct_nkxpad)*ct_ny+yid];
        else                      out[tid] = CZERO;
    }
}

void update_shear
    ( const cureal delt 
){
    dim3 block( nthread );
    dim3 kxgrid( ((nkx-1)+nthread-1)/nthread );
    dim3 cgrid( (nkx*nky+nthread-1)/nthread );

    shearing_ky <<< kxgrid, block >>> ( delt );
    get_kperp2_shear <<< cgrid, block >>> ();

    hipMemcpy( &jump_flag, dv_jump+1, sizeof(int), hipMemcpyDeviceToHost );
    if( jump_flag != 0 ){
        hipMemcpy( dv_ctmp1, dv_aomg0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_aomg0 );
        hipMemcpy( dv_ctmp1, dv_aomg1, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_aomg1 );
        hipMemcpy( dv_ctmp1, dv_aomg2, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_aomg2 );

        hipMemcpy( dv_ctmp1, dv_domg0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_domg0 );
        hipMemcpy( dv_ctmp1, dv_domg1, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_domg1 );
        hipMemcpy( dv_ctmp1, dv_domg2, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_domg2 );

        hipMemcpy( dv_ctmp1, dv_arho0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_arho0 );
        hipMemcpy( dv_ctmp1, dv_arho1, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_arho1 );
        hipMemcpy( dv_ctmp1, dv_arho2, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_arho2 );

        hipMemcpy( dv_ctmp1, dv_drho0, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_drho0 );
        hipMemcpy( dv_ctmp1, dv_drho1, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_drho1 );
        hipMemcpy( dv_ctmp1, dv_drho2, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_drho2 );

        hipMemcpy( dv_ctmp1, dv_aphi, sizeof(cucmplx)*nkx*nky, hipMemcpyDeviceToDevice );
        shearing_field <<< cgrid, block >>> ( dv_ctmp1, dv_aphi );
    }
}

__global__ static void shearing_ky
    ( const cureal delt
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if( tid <= ct_nkx-1 ){
        gb_ky_shift[tid] = gb_ky_shift[tid] - ct_sigma * gb_kx[tid] * delt;
        gb_jump[tid] = floor( gb_ky_shift[tid]/gb_ky[1] + 0.5 );

        __syncthreads();

        if( gb_jump[1] != 0 )
            gb_ky_shift[tid] = gb_ky_shift[tid] - gb_jump[tid] * gb_ky[1];
    }
}

__global__ static void get_kperp2_shear
    ( void
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_nky, yid = tid%ct_nky;
    cureal kxs, kys;

    if( tid < ct_nkx*ct_nky ){
        kxs = gb_kx[xid];
        kys = gb_ky[yid] + gb_ky_shift[xid];
        gb_kperp2_shear[tid] = kxs*kxs + kys*kys;
    }
}

__global__ static void shearing_field
    ( const cucmplx *in
    ,       cucmplx *out
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_nky, yid = tid%ct_nky;

    if( tid < ct_nkx*ct_nky ){
        if( gb_jump[xid] <= 0 ){
            if( yid-gb_jump[xid] < ct_nky ){
                out[tid] = in[xid*ct_nky+(yid-gb_jump[xid])];
            }
            else{
                out[tid] = CZERO;
            }
        }
        else{
            if( yid-gb_jump[xid] > 0 ){
                out[tid] = in[xid*ct_nky+(yid-gb_jump[xid])];
            }
            else{
                out[tid].x =  in[gb_ikx_indexed[xid]*ct_nky+(gb_jump[xid]-yid)].x;
                out[tid].y = -in[gb_ikx_indexed[xid]*ct_nky+(gb_jump[xid]-yid)].y;
            }
        }
    }
}
