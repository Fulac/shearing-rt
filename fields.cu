#include <hip/hip_runtime.h>
#include "cmplx.h"
#include "fft.h"

cureal delt, tmax;
cureal cfl_vx, cfl_vy;
cureal nu, kappa, sigma, rho0, rho0_prime, g, rho_eps1, rho_eps2, eps;
__constant__ cureal ct_nu, ct_sigma, ct_rho0, ct_rho0_prime, ct_g, ct_kappa, ct_rho_eps2;
cureal dx, dy;

cureal *xx, *yy, *omgz, *phi, *rho;
cureal *dv_xx, *dv_yy, *dv_omgz, *dv_phi, *dv_rho, *dv_vx, *dv_vy;
__device__ cureal *gb_xx, *gb_yy;
cucmplx *dv_aomg0, *dv_aomg1, *dv_aomg2;
cucmplx *dv_domg0, *dv_domg1, *dv_domg2;
cucmplx *dv_arho0, *dv_arho1, *dv_arho2;
cucmplx *dv_drho0, *dv_drho1, *dv_drho2;
cucmplx *dv_aphi;

/* ---------------------------------------------------------------- */

void init_fields( void );
void finish_fields( void );

static void allocate_CPU( void );
static void deallocate_CPU( void );

static void allocate_GPU( void );
static void deallocate_GPU( void );

/* ---------------------------------------------------------------- */

void init_fields
    ( void 
){
    hipMemcpyToSymbol(HIP_SYMBOL( ct_nu),         &nu,         sizeof(cureal) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_sigma),      &sigma,      sizeof(cureal) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_rho0),       &rho0,       sizeof(cureal) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_rho0_prime), &rho0_prime, sizeof(cureal) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_g),          &g,          sizeof(cureal) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_kappa),      &kappa,      sizeof(cureal) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_rho_eps2),   &rho_eps2,   sizeof(cureal) );

    allocate_CPU();
    allocate_GPU();
}

void finish_fields
    ( void 
){
    deallocate_CPU();
    deallocate_GPU();
}

static void allocate_CPU
    ( void 
){
    int nn = nx * ny;

    xx = new cureal [nx+1];
    yy = new cureal [ny+1];
    omgz = new cureal [nn];
    phi = new cureal [nn];
    rho = new cureal [nn];
}

static void deallocate_CPU
    ( void 
){
    delete[] xx;
    delete[] yy;
    delete[] omgz;
    delete[] phi;
    delete[] rho;
}

static void allocate_GPU
    ( void 
){
    int nn;

    hipMalloc( (void**)&dv_xx, sizeof(cureal) * (nx+1) );
    hipMalloc( (void**)&dv_yy, sizeof(cureal) * (ny+1) );

    nn = nx * ny;
    hipMalloc( (void**)&dv_vx, sizeof(cureal) * nn );
    hipMemset( dv_vx, 0, sizeof(cureal) * nn );
    hipMalloc( (void**)&dv_vy, sizeof(cureal) * nn );
    hipMemset( dv_vy, 0, sizeof(cureal) * nn );
    hipMalloc( (void**)&dv_omgz, sizeof(cureal) * nn );
    hipMalloc( (void**)&dv_phi, sizeof(cureal) * nn );
    hipMalloc( (void**)&dv_rho, sizeof(cureal) * nn );

    nn = nkx * nky;
    hipMalloc( (void**)&dv_aomg0, sizeof(cucmplx) * nn );
    hipMemset( dv_aomg0, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_aomg1, sizeof(cucmplx) * nn );
    hipMemset( dv_aomg1, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_aomg2, sizeof(cucmplx) * nn );
    hipMemset( dv_aomg2, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_domg0, sizeof(cucmplx) * nn );
    hipMemset( dv_domg0, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_domg1, sizeof(cucmplx) * nn );
    hipMemset( dv_domg1, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_domg2, sizeof(cucmplx) * nn );
    hipMemset( dv_domg2, 0, sizeof(cucmplx) * nn );

    hipMalloc( (void**)&dv_arho0, sizeof(cucmplx) * nn );
    hipMemset( dv_aomg0, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_arho1, sizeof(cucmplx) * nn );
    hipMemset( dv_aomg1, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_arho2, sizeof(cucmplx) * nn );
    hipMemset( dv_aomg2, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_drho0, sizeof(cucmplx) * nn );
    hipMemset( dv_domg0, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_drho1, sizeof(cucmplx) * nn );
    hipMemset( dv_domg1, 0, sizeof(cucmplx) * nn );
    hipMalloc( (void**)&dv_drho2, sizeof(cucmplx) * nn );
    hipMemset( dv_domg2, 0, sizeof(cucmplx) * nn );

    hipMalloc( (void**)&dv_aphi, sizeof(cucmplx) * nn );
    hipMemset( dv_aphi, 0, sizeof(cucmplx) * nn );
}

static void deallocate_GPU
    ( void 
){
    hipFree( dv_xx );
    hipFree( dv_yy );
    hipFree( dv_vx );
    hipFree( dv_vy );
    hipFree( dv_omgz );
    hipFree( dv_phi );

    hipFree( dv_aomg0 );
    hipFree( dv_aomg1 );
    hipFree( dv_aomg2 );
    hipFree( dv_domg0 );
    hipFree( dv_domg1 );
    hipFree( dv_domg2 );

    hipFree( dv_arho0 );
    hipFree( dv_arho1 );
    hipFree( dv_arho2 );
    hipFree( dv_drho0 );
    hipFree( dv_drho1 );
    hipFree( dv_drho2 );

    hipFree( dv_aphi );
}
