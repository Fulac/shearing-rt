#include "hip/hip_runtime.h"
// CUDA headers
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Local headers
#include "cmplx.h"

/* ---------------------------------------------------------------------------------------------- */
/*  Global Variables Definition                                                                   */
/* ---------------------------------------------------------------------------------------------- */

// プログラム全体で使用する変数を定義
int nthread;
int nx, ny, nkx, nky, nkxh, nkxh2, nkxpad, ncy;
__constant__ int ct_nx, ct_ny, ct_nkx, ct_nky;
__constant__ int ct_nkxh, ct_nkxh2, ct_nkxpad, ct_ncy;

// このファイル内でのみ使用するグローバル変数を定義
namespace{
    cureal  *dv_rtmp;
    cucmplx *dv_ctmp1, *dv_ctmp2;

    hipfftHandle pr2c, pc2r, pc2c;
}

////////////////////////////////////////////////////////////////////////////////////////////////////


/* ---------------------------------------------------------------------------------------------- */
/*  Function Prototype                                                                            */
/* ---------------------------------------------------------------------------------------------- */

void init_fft
    ( void
);
void finish_fft
    ( void
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void xtok
    ( cureal  *in
    , cucmplx *out
);

__global__ static void scale_dealias
    ( const cucmplx *in
    ,       cucmplx *out
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void ktox
    ( const cucmplx *in
    ,       cureal  *out
);

__global__ static void pad2d
    ( const cucmplx *in
    ,       cucmplx *out
);

////////////////////////////////////////////////////////////////////////////////////////////////////

void ktox_1d
    ( const cucmplx *in
    ,       cureal  *out
);

__global__ static void transpose
    ( const cucmplx *in
    ,       cucmplx *out
);

__global__ static void trans_inv
    ( const cucmplx *in
    ,       cureal  *out
);

////////////////////////////////////////////////////////////////////////////////////////////////////


/* ---------------------------------------------------------------------------------------------- */
/*  Function Definition                                                                           */
/* ---------------------------------------------------------------------------------------------- */

void init_fft
    ( void 
){
    nkx    = (nx-1)/3*2 + 1;
    nky    = (ny-1)/3 + 1;
    nkxh   = (nkx+1)/2;
    nkxh2  = nx - nkxh;
    nkxpad = nx - nkxh*2 + 1;
    ncy    = ny/2 + 1;

    hipMemcpyToSymbol(HIP_SYMBOL( ct_nx),     &nx,     sizeof(int) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_ny),     &ny,     sizeof(int) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_nkx),    &nkx,    sizeof(int) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_nky),    &nky,    sizeof(int) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_nkxh),   &nkxh,   sizeof(int) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_nkxh2),  &nkxh2,  sizeof(int) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_nkxpad), &nkxpad, sizeof(int) );
    hipMemcpyToSymbol(HIP_SYMBOL( ct_ncy),    &ncy,    sizeof(int) );

    hipMalloc( (void**)&dv_rtmp,  sizeof(cureal)  * nx * ny  );
    hipMalloc( (void**)&dv_ctmp1, sizeof(cucmplx) * nx * ncy );
    hipMalloc( (void**)&dv_ctmp2, sizeof(cucmplx) * nx * ny  );

    #ifdef DBLE
        hipfftPlan2d( &pr2c, nx, ny, HIPFFT_D2Z );
        hipfftPlan2d( &pc2r, nx, ny, HIPFFT_Z2D );
        hipfftPlan1d( &pc2c, nx, HIPFFT_Z2Z, 1  );
    #else
        hipfftPlan2d( &pr2c, nx, ny, HIPFFT_R2C );
        hipfftPlan2d( &pc2r, nx, ny, HIPFFT_C2R );
        hipfftPlan1d( &pc2c, nx, HIPFFT_C2C, 1  );
    #endif
}

void finish_fft
    ( void 
){
    hipfftDestroy( pr2c );
    hipfftDestroy( pc2r );
    hipfftDestroy( pc2c );

    hipFree( dv_rtmp  );
    hipFree( dv_ctmp1 );
    hipFree( dv_ctmp2 );
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void xtok
    ( cureal  *in
    , cucmplx *out 
){
    dim3 block( nthread );
    dim3 cgrid( (nkx*nky+nthread-1)/nthread );

    hipMemset( dv_ctmp1, 0, sizeof(cucmplx) * nx * ncy );

    hipMemcpy( dv_rtmp, in, sizeof(cureal) * nx * ny, hipMemcpyDeviceToDevice );
    #ifdef DBLE
        hipfftExecD2Z( pr2c, dv_rtmp, dv_ctmp1 );
    #else
        hipfftExecR2C( pr2c, dv_rtmp, dv_ctmp1 );
    #endif

    scale_dealias <<< cgrid, block >>> ( dv_ctmp1, out );
}

__global__ static void scale_dealias
    ( const cucmplx *in
    ,       cucmplx *out 
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_nky, yid = tid%ct_nky;

    if( tid < ct_nkx*ct_nky ){
        if( xid < ct_nkxh ) out[tid] = in[xid*ct_ncy+yid] / (ct_nx*ct_ny);
        else out[tid] = in[(xid+ct_nkxpad)*ct_ncy+yid] / (ct_nx*ct_ny);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void ktox
    ( const cucmplx *in
    ,       cureal  *out 
){
    dim3 block( nthread );
    dim3 rgrid( (nx*ny+nthread-1)/nthread );
    dim3 rcgrid( (nx*ncy+nthread-1)/nthread );

    hipMemset( dv_ctmp1, 0, sizeof(cucmplx) * nx * ncy );

    pad2d <<< rcgrid, block >>> ( in, dv_ctmp1 );

    #ifdef DBLE
        hipfftExecZ2D( pc2r, dv_ctmp1, out );
    #else
        hipfftExecC2R( pc2r, dv_ctmp1, out );
    #endif
}

__global__ static void pad2d
    ( const cucmplx *in
    ,       cucmplx *out 
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_ncy, yid = tid%ct_ncy;

    if( tid < ct_nx*ct_ncy ){
        if( yid < ct_nky ){
            if( xid < ct_nkxh ) out[tid] = in[xid*ct_nky+yid];
            else if( xid > ct_nkxh2 ) out[tid] = in[(xid-ct_nkxpad)*ct_nky+yid];
            else out[tid] = CZERO;
        }
        else out[tid] = CZERO;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void ktox_1d
    ( const cucmplx *in
    ,       cureal  *out
){
    dim3 block( nthread );
    dim3 rgrid( (nx*ny+nthread-1)/nthread );

    transpose <<< rgrid, block >>> ( in, dv_ctmp2 );

    for( int iy = 0; iy < ny; iy++ ){
        #ifdef DBLE
            hipfftExecZ2Z( pc2c, dv_ctmp2+iy*nx, dv_ctmp2+iy*nx, HIPFFT_BACKWARD );
        #else
            hipfftExecC2C( pc2c, dv_ctmp2+iy*nx, dv_ctmp2+iy*nx, HIPFFT_BACKWARD );
        #endif
    }

    trans_inv <<< rgrid, block >>> ( dv_ctmp2, out );
}

__global__ static void transpose
    ( const cucmplx *in
    ,       cucmplx *out 
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid/ct_ny, yid = tid%ct_ny;

    if( tid < ct_nx*ct_ny ){
        out[yid*ct_nx+xid] = in[tid];
    }
}

__global__ static void trans_inv
    ( const cucmplx *in
    ,       cureal  *out 
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int xid = tid%ct_nx, yid = tid/ct_nx;

    if( tid < ct_nx*ct_ny ){
        out[xid*ct_ny+yid] = in[tid].x;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
